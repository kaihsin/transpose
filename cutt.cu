#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutt.h>
#include <random>
#include <complex>

typedef float floatType;

#define CudaSafeCall(err) __cudaSafeCall( err, __FILE__, __LINE__ )
void __cudaSafeCall(hipError_t err, const char *file, const int line);

void __cudaSafeCall(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
}

void cuttCheck(cuttResult err) {
  if (CUTT_SUCCESS != err) {
    fprintf(stderr, "cuttCheck() failed at %s:%i\n", __FILE__, __LINE__);
    exit(-1);
  }
}

const int NUM_REPS = 10;


void parse(char *source, std::vector<int>& vec)
{
  char delims[] = "=,";
  char *token;
  
  token = strtok(source, delims); // remove flag
  token = strtok(NULL, delims);
  while (token != NULL) {
    //printf("%s\n", token);
    vec.push_back(atoi(token));
    token = strtok(NULL, delims);
  }
}

int main(int argc, char *argv[])
{
  int bondDim;
  char* datname;
  std::vector<int> vecperm;
  std::vector<int> vecsize;
  for(int i = 0; i < argc; ++i) {
    if (!strncmp(argv[i], "--bond", 6)) {
      char delims[] = "=,";
      char *token;
      token = strtok(argv[i], delims); // remove flag
      token = strtok(NULL, delims);
      bondDim = atoi(token);
    }
    if (!strncmp(argv[i], "--datname", 9)) {
      char delims[] = "=";
      char *token;
      token = strtok(argv[i], delims); // remove flag
      token = strtok(NULL, delims);
      datname = token;
    }
    if (!strncmp(argv[i], "--size", 6))
      parse(argv[i], vecsize);
    if (!strncmp(argv[i], "--perm", 6))
      parse(argv[i], vecperm);
  }

  int dim = vecsize.size();
  int* perm = vecperm.data();
  int* size = vecsize.data();

  size_t total_size = 1;
  for(int i = 0; i < dim ; ++i)
    total_size *= size[i];

  int memSize = total_size*sizeof(floatType);

  floatType *h_idata = (floatType*)malloc(memSize);
  floatType *h_odata = (floatType*)malloc(memSize);

  floatType *d_idata, *d_odata;
  CudaSafeCall( hipMalloc(&d_idata, memSize) );
  CudaSafeCall( hipMalloc(&d_odata, memSize) );

  for (int i = 0; i < total_size; i++)
    h_idata[i] = (floatType)rand() / RAND_MAX;

  CudaSafeCall( hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice) );
  CudaSafeCall( hipMemset(d_odata, 0, memSize) );


  // Events for timing
  hipEvent_t startEvent, stopEvent;
  CudaSafeCall( hipEventCreate(&startEvent) );
  CudaSafeCall( hipEventCreate(&stopEvent) );
  float ms;

  cuttHandle plan;
  cuttCheck( cuttPlan(&plan, dim, vecsize.data(), vecperm.data(), sizeof(floatType), 0) );

  double minTime = 1e100;
  for (int i = 0; i < NUM_REPS; i++) {
    CudaSafeCall( hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice) );

    CudaSafeCall( hipEventRecord(startEvent, 0) );

    cuttCheck( cuttExecute(plan, d_idata, d_odata) );

    CudaSafeCall( hipEventRecord(stopEvent, 0) );
    CudaSafeCall( hipEventSynchronize(stopEvent) );
    CudaSafeCall( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    minTime = (minTime < ms) ? minTime : ms;
  }

  FILE *pf = fopen(datname, "a");
  fprintf(pf,"%5d%20.12f%20.12f\n", bondDim, ms, 2*total_size*sizeof(float)*1e-6/ms);
  fclose(pf);

  cuttCheck( cuttDestroy(plan) );
}
